/* From: https://github.com/mvx24 
 * Find the sum of all primes below 2 million (Project Euler #10).
 * This can take a while! *spoiler* 142913828922
 * For below 2k: 277050 (0.09s via nvcc, 19 hours via kcc!)
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK     512
#define START_NUMBER          1414
#define TOTAL_THREADS         ((2002-START_NUMBER)/2)

// Kernel that executes on the CUDA device
__global__ void sum_primes(int* firstPrimes, size_t n, unsigned long long* blockSums) {
      __shared__ int blockPrimes[THREADS_PER_BLOCK];
      int i;
      int idx;
      int num;

      idx = blockIdx.x * blockDim.x + threadIdx.x;
      if (idx < TOTAL_THREADS) {
            // The number to test
            num = (START_NUMBER - 1) + (idx * 2);
            for (i = 0; i < n; ++i) {
                  if(!(num % firstPrimes[i])) break;
            }
            if (i == n)
                  blockPrimes[threadIdx.x] = num;
            else
                  blockPrimes[threadIdx.x] = 0;
      } else {
            blockPrimes[threadIdx.x] = 0;
      }

      __syncthreads();

      if (threadIdx.x == 0) {
            // sum all the results from the block
            blockSums[blockIdx.x] = 0;
            for (i = 0; i < blockDim.x; ++i)
                  blockSums[blockIdx.x] += blockPrimes[i];
      }
}

// main routine that executes on the host
int main(int argc, char *argv[]) {
      //host
      int primes[1024];
      unsigned long long *primeSums;
      int i, j, index;
      int blockSize, nblocks;
      unsigned long long sum;
      size_t len;

      //device
      int* primesDevice;
      unsigned long long* primeSumsDevice;

      // Find all the primes less than the square root of 2 million ~1414
      primes[0] = 2;
      index = 1;
      sum = 2;
      for (i = 3; i != START_NUMBER; ++i) {
            for (j = 0; j != index; ++j) {
                  if (!(i % primes[j])) break;
            }
            if (j == index) {
                  primes[index++] = i;
                  sum += i;
            }
      }
      len = index;

      hipMalloc((void**) &primesDevice, len * sizeof(int));
      hipMemcpy(primesDevice, primes, len * sizeof(int), hipMemcpyHostToDevice);

      // Test the all odd numbers between 1414 and 2000000
      blockSize = THREADS_PER_BLOCK;
      nblocks = TOTAL_THREADS/blockSize + !!(TOTAL_THREADS % blockSize);
      hipMalloc((void**) &primeSumsDevice, nblocks * sizeof(unsigned long long));

      sum_primes <<< nblocks, blockSize >>> (primesDevice, index, primeSumsDevice);
      // C invocation
      // do {
      //       dim3 gridDim;
      //       dim3 blockDim;
      //       cudaError_t error;
      //       gridDim.x = nblocks;
      //       blockDim.x = blockSize;
      //       gridDim.y = gridDim.z = blockDim.y = blockDim.z = 1;
      //       error = cudaConfigureCall(gridDim, blockDim, 0, NULL);
      //       if(error != cudaSuccess)
      //       {
      //             printf("%s\n", cudaGetErrorString(error));
      //             break;
      //       }
      //       error = cudaSetupArgument(&primesDevice, sizeof(primesDevice), 0);
      //       error = cudaSetupArgument(&index, sizeof(index), sizeof(primesDevice));
      //       error = cudaSetupArgument(&primeSumsDevice, sizeof(primeSumsDevice), sizeof(primesDevice) + sizeof(index));
      //       printf("Start kernel\n");
      //       error = cudaLaunch(sum_primes);
      //       if(error != cudaSuccess) {
      //             printf("cudaLaunch: %s\n", cudaGetErrorString(error));
      //             break;
      //       }
      // } while(0);

      // Retrieve result from device and store it in host array
      primeSums = (unsigned long long*) malloc(nblocks * sizeof(unsigned long long));
      hipMemcpy(primeSums, primeSumsDevice, nblocks * sizeof(unsigned long long), hipMemcpyDeviceToHost);
      for (i = 0; i != nblocks; ++i) {
            sum += primeSums[i];
            //printf("%llu\t", primeSums[i]);
      }

      // Cleanup
      free(primeSums);
      hipFree(primeSumsDevice);
      hipFree(primesDevice);

      // Print results
      printf("%llu\n", sum);
}