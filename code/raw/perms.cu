
#include <hip/hip_runtime.h>
#define NUMBER_OF_ELEMENTS 5
#define BLOCK_DIM 1024
#define OFFSET 0
// When MAX_PERM = 0, means find all permutations
#define MAX_PERM 0
#define NEXT_PERM_LOOP 1

__constant__ long long arr[20][20] = { /*Not shown here to save space*/ };

// function to swap character 
// a - the character to swap with b
// b - the character to swap with a
__device__ void swap(
    char* a, 
    char* b)
{
    char tmp = *a;
    *a = *b;
    *b = tmp;
}


// function to reverse the array (sub array in array)
// first - 1st character in the array (sub-array in array)
// last - 1 character past the last character
__device__ void reverse(
    char* first, 
    char* last)
{    
    for (; first != last && first != --last; ++first)
        swap(first, last);
}


// function to find the next permutation (sub array in array)
// first - 1st character in the array (sub-array in array)
// last - 1 character past the last character
__device__ void next_permutation(
    char* first, 
    char* last)
{
    char* next = last;
    --next;
    if(first == last || first == next)
        return;

    while(true)
    {
        char* next1 = next;
        --next;
        if(*next < *next1)
        {
            char* mid = last;
            --mid;
            for(; !(*next < *mid); --mid)
                ;
            swap(next, mid);
            reverse(next1, last);
            return;
        }

        if(next == first)
        {
            reverse(first, last);
            return;
        }
    }
}    

__global__ void PermuteHybrid(char* arrDest, long long* offset, long long* Max)
{
    long long index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= (*Max/(NEXT_PERM_LOOP+1)))
        return;

    index *= NEXT_PERM_LOOP+1;
    long long tmpindex = index;
    
    index += *offset;
    
    char arrSrc[NUMBER_OF_ELEMENTS];
    char arrTaken[NUMBER_OF_ELEMENTS];
    for(char i=0; i<NUMBER_OF_ELEMENTS; ++i)
    {
        arrSrc[i] = i;
        arrTaken[i] = 0;
    }

    char size = NUMBER_OF_ELEMENTS;
    for(char i=NUMBER_OF_ELEMENTS-1; i>=0; --i)
    {
        for(char j=i; j>=0; --j)
        {
            if(index >= arr[i][j])
            {
                char foundcnt = 0;
                index = index - arr[i][j];
                for(char k=0;k<NUMBER_OF_ELEMENTS; ++k)
                {
                    if(arrTaken[k]==0) // not taken
                    {
                        if(foundcnt==j)
                        {
                            arrTaken[k] = 1; // set to taken
                            arrDest[ (tmpindex*NUMBER_OF_ELEMENTS) + (NUMBER_OF_ELEMENTS-size) ] = arrSrc[k];
                            break;
                        }
                        foundcnt++;
                    }
                }
                break;
            }
        }
        --size;
    }

    long long idx = tmpindex*NUMBER_OF_ELEMENTS;
    for(char a=1; a<NEXT_PERM_LOOP+1; ++a)
    {
        long long idx2 = a*NUMBER_OF_ELEMENTS;
        for(char i=0; i<NUMBER_OF_ELEMENTS; ++i)
        {
            arrDest[ idx + idx2 + i ] =
                arrDest[ idx + ((a-1)*NUMBER_OF_ELEMENTS) + i ];
        }
        next_permutation(arrDest + idx + idx2, 
            arrDest+idx + idx2 + NUMBER_OF_ELEMENTS);
    }
}